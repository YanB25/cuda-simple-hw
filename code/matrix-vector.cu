#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define INDEX(fst, snd, n) ((fst) * (n) + (snd))
#define SIZE (5000)
#define TILL (100)
#define N_TILL (SIZE / TILL)
__global__ void multiple(float* matrix, float* vector, float* out) {
    /*
     * a thread get 100 element in a line (one line 50 thread)
     * thread is (50, 20)
     * 20 lines fill a block.
     * a matrix has 250 blocks
     */
    int x = threadIdx.x;
    int y = threadIdx.y;
    int blk = blockIdx.x;
    float sum = 0;
    for (int i = x * 100; i < (x + 1) * 100; ++i) {
        sum += matrix[INDEX(y, i)] * vector[i];
    }
    out[y + blk * 20] = sum;
}
int main() {
    float* hA = (float*) malloc(sizeof(float) * SIZE * SIZE);
    float* dA;
    hipMalloc((void**) &dA, sizeof(float) * SIZE * SIZE);
    float* hx = (float*) malloc(sizeof(float) * SIZE);
    float* dx;
    hipMalloc((void**) &dx, sizeof(float) * SIZE * SIZE);
    float* out;
    hipMalloc((void**) &out, sizeof(float) * SIZE);

    // init hA and hx
    for (int i = 0; i < SIZE; ++i) {
        for (int j = 0; j < SIZE; ++j) {
            hA[INDEX(i, j)] = i - 0.1 * j + 1;
        }
        hx[i] = 0.2 * i - 0.1 * sqrt(i);
    }

    // init out
    hipMemset(out, 0, sizeof(float)* SIZE);

    // transfer to gpu
    hipMemcpy(dA, hA, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dx, hx, sizeof(float) * SIZE, hipMemcpyHostToDevice);

    dim3 threads(50, 20);
    multiple<<<250, threads>>>();

    free(hA);
    free(hx);
    hipFree(dA);
    hipFree(dx);
    float* hout = (float*) malloc(sizeof(float) * SIZE);
    hipMemcpy(hout, out, sizeof(float)* SIZE, hipMemcpyDeviceToHost);
    for (int i = 0; i < 100; ++i) {
        printf("%f\n", hout[i]);
    }
    free(hout);
    hipFree(out);
}