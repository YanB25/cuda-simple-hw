
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define DEBUG

#define INDEX(fst, snd, n) ((fst) * (n) + (snd))
#define SIZE (5000)
#define TILL (100)
#define N_TILL (SIZE / TILL)
__global__ void multiple(float* matrix, float* vector, float* out) {
    /*
     * a thread get 100 element in a line (one line 50 thread)
     * thread is (50, 20)
     * 20 lines fill a block.
     * a matrix has 250 blocks
     */
    int x = threadIdx.x;
    int y = threadIdx.y;
    int blk = blockIdx.x;
    float sum = 0;
    for (int i = x * 100; i < (x + 1) * 100; ++i) {
        sum += matrix[INDEX(y + blk * 20, i, SIZE)] * vector[i];
    }
    atomicAdd(&out[y + blk * 20], (float)sum);
}

void validator(float* matrix, float* vector, float* out) {
    for (int i = 0; i < SIZE; ++i) {
        float sum = 0;
        for (int j = 0; j < SIZE; ++j) {
            sum += matrix[INDEX(i, j, SIZE)] * vector[j];
        }
        out[i] = sum;
    }
}


int main() {
    float* hA = (float*) malloc(sizeof(float) * SIZE * SIZE);
    float* dA;
    hipMalloc((void**) &dA, sizeof(float) * SIZE * SIZE);
    float* hx = (float*) malloc(sizeof(float) * SIZE);
    float* dx;
    hipMalloc((void**) &dx, sizeof(float) * SIZE * SIZE);
    float* out;
    hipMalloc((void**) &out, sizeof(float) * SIZE);
    float* valout = (float*) malloc(sizeof(float)  * SIZE);

    // init hA and hx
    for (int i = 0; i < SIZE; ++i) {
        for (int j = 0; j < SIZE; ++j) {
            hA[INDEX(i, j, SIZE)] = i - 0.1 * j + 1;
        }
        hx[i] = 0.2 * i - 0.1 * sqrt(i);
    }

    // init out
    hipMemset(out, 0, sizeof(float)* SIZE);
    memset(valout, 0, sizeof(float) * SIZE);

    // transfer to gpu
    hipMemcpy(dA, hA, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dx, hx, sizeof(float) * SIZE, hipMemcpyHostToDevice);

    dim3 threads(50, 20);
    multiple<<<250, threads>>>(dA, dx, out);
    validator(hA, hx, valout);

    free(hA);
    free(hx);
    hipFree(dA);
    hipFree(dx);
    float* hout = (float*) malloc(sizeof(float) * SIZE);
    hipMemcpy(hout, out, sizeof(float)* SIZE, hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; ++i) {
        printf("%f, (%f) \n", hout[i], valout[i]);
    }
    free(valout);
    free(hout);
    hipFree(out);
}

