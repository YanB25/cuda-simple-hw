
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void helloCUDA() {
    printf("Hello from thread (%d, %d) block (%d, %d)\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
}
int main() {
    dim3 grid(2, 4);
    dim3 block(8, 16);
    helloCUDA<<<grid, block>>>();
    hipDeviceSynchronize();
    return 0;
}
