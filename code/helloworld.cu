#include "hip/hip_runtime.h"
#include <stdio.h>
__global__ void helloCUDA(float f) {
    printf(("Hello thread %d, f=%f\n", threadIdx.x, f);
}
int main() {
    dim3 grid(2, 4);
    dim3 block(8, 16);
    helloCUDA<<<grid, block>>>(1.23456f);
    return 0;
}